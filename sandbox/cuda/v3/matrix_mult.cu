#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include "matrix_sizes.h"
#include "matrix_mult.h"

// CUDA Kernel
// Multiply two matrices A * B = C
__global__ void
cudakernel_matrix_mul( float* C, float* A, float* B, int wA, int wB)
{
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed
  // by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed
  // by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the
  // sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed
  // by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the
  // sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  float Csub = 0.;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
           a <= aEnd;
           a += aStep, b += bStep)
  {

    // Declaration of the shared memory array As
    // used to store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs
    // used to store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from global memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices
    // are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    Csub = 0.;
    for (int k = 0; k < BLOCK_SIZE; ++k)
      Csub += As[ty][k] * Bs[k][tx];

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

  
void gpu_mat_mul(float* h_A, float* h_B, float* h_C )
{

    // allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;

    unsigned int size_A = WA * HA;
    unsigned int size_B = WB * HB;
    unsigned int size_C = WC * HC;

    unsigned int mem_size_A = sizeof(float) * size_A;
    unsigned int mem_size_B = sizeof(float) * size_B;
    unsigned int mem_size_C = sizeof(float) * size_C;

    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
    hipMalloc((void**) &d_C, mem_size_C);

    // copy host memory to device*/
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
 
    // perform the calculation

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);
 
    //   execute the kernel
    cudakernel_matrix_mul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
 
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
