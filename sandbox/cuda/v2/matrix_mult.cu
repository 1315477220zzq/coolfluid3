#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include "matrix_sizes.h"
#include "matrix_mult.h"

#if 0 // version 1
// CUDA Kernel
__global__ void
cudakernel_matrix_mul( float* C, float* A, float* B, int wA, int wB)
{

   // 2D Thread ID
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   // value stores the element that is
   // computed by the thread
   float value = 0;
   for (int i = 0; i < wA; ++i)
   {
      float elementA = A[ty * wA + i];
      float elementB = B[i * wB + tx];
      value += elementA * elementB;
   }

   // Write the matrix to device memory each
   // thread writes one element
   C[ty * wA + tx] = value;
}
#endif

#if 1 // version 2
// CUDA Kernel
// Multiply two matrices A * B = C
__global__ void
cudakernel_matrix_mul( float* C, float* A, float* B, int wA, int wB)
{

   // 2D Thread ID
   int tx = blockIdx.x * TILE_SIZE + threadIdx.x;
   int ty = blockIdx.y * TILE_SIZE + threadIdx.y;

   // value stores the element that is
   // computed by the thread
   float value = 0;
   for (int i = 0; i < wA; ++i)
   {
     float elementA = A[ty * wA + i];
     float elementB = B[i * wB + tx];
      value += elementA * elementB;
   }

   // Write the matrix to device memory
   // each thread writes one element
   C[ty * wA + tx] = value;
}
#endif

void gpu_mat_mul(float* h_A, float* h_B, float* h_C )
{

    // allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;

    unsigned int size_A = WA * HA;
    unsigned int size_B = WB * HB;
    unsigned int size_C = WC * HC;

    unsigned int mem_size_A = sizeof(float) * size_A;
    unsigned int mem_size_B = sizeof(float) * size_B;
    unsigned int mem_size_C = sizeof(float) * size_C;

    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
    hipMalloc((void**) &d_C, mem_size_C);

    // copy host memory to device*/
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
 
    // perform the calculation

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);
 
    //   execute the kernel
    cudakernel_matrix_mul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
 
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
