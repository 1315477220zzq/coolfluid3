
/* Multiply two matrices A * B = C */
 
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include "matrix_sizes.h"
#include "matrix_mult_kernel.h"

/* Allocates a matrix with random float entries.*/
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
  
int
main(int argc, char** argv)
{

    /* set seed for rand()*/
    srand(2006);
 
    /* 1. allocate host memory for matrices A and B*/
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
 
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
 
    /* 2. initialize host memory*/
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
  
    /* 3. print out A and B*/
    printf("\n\nMatrix A\n");
    for( unsigned int i = 0; i < size_A; i++)
    {
       printf("%f ", h_A[i]);
       if(((i + 1) % WA) == 0)
          printf("\n");
    }
 
    printf("\n\nMatrix B\n");
    for( unsigned int i = 0; i < size_B; i++)
    {
       printf("%f ", h_B[i]);
       if(((i + 1) % WB) == 0)
          printf("\n");
    }
 
    /* 8. allocate device memory*/
    float* d_A;
    float* d_B;
    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
 
    /* 9. copy host memory to device*/
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

 
    /* 4. allocate host memory for the result C*/
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* h_C = (float*) malloc(mem_size_C);
 
    /* 10. allocate device memory for the result*/
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);
 
    /* 5. perform the calculation    */
    /*    setup execution parameters */
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);
 
    /*   execute the kernel */
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
 
    /* 11. copy result from device to host */
    hipMemcpy(h_C, d_C, mem_size_C,
               hipMemcpyDeviceToHost);
 
    /* 6. print out the results */
    printf("\n\nMatrix C (Results)\n");
    for( unsigned  int i = 0; i < size_C; i++)
    {
       printf("%f ", h_C[i]);
       if(((i + 1) % WC) == 0)
          printf("\n");
    }
    printf("\n");
 
    /* 7. clean up memory */
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
